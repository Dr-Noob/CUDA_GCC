#include <hip/hip_runtime.h>

__global__
void vectorAdd(float *idata, float *odata, int size) {
  /* really interesting computation */  
}

void my_wrap(int blocksPerGrid, int threadsPerBlock, float *idata, float *odata, int size) {
  vectorAdd<<<blocksPerGrid, threadsPerBlock, 0>>>(idata, odata, size);
}
